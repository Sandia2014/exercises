#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <cstdio>

#include <hip/hip_runtime.h>

#include "ex1_histogram_cuda.cuh"

__global__
void
cudaDoHistogramPopulation_kernel(unsigned int * d_input, unsigned int * d_output,
                                unsigned int numElements,
                                unsigned int bucketSize) {
  unsigned int myID = (blockIdx.x * blockDim.x) + threadIdx.x;

  if(myID < numElements) {
    const unsigned int value = d_input[myID];
    const unsigned int bucketNumber = value / bucketSize;
    atomicAdd(&(d_output[bucketNumber]), (int) 1);
  }
}

void
cudaDoHistogramPopulation(const unsigned int threadsPerBlock,
                          unsigned int * h_outputHistogram,
                          unsigned int * h_cudaInput,
                          unsigned int numElements,
                          unsigned int numBuckets) {

    unsigned int * d_input;
    unsigned int * d_output;
    hipMalloc(&d_input, sizeof(unsigned int) * numElements);
    hipMalloc(&d_output, sizeof(unsigned int) * numBuckets);
    hipMemset(d_output, 0, sizeof(unsigned int) * numBuckets);

    hipMemcpy(d_input, h_cudaInput,
            sizeof(unsigned int) * numElements, hipMemcpyHostToDevice);

    dim3 blockSize(threadsPerBlock);
    dim3 gridSize((numElements / threadsPerBlock) + 1);
    const unsigned int bucketSize = numElements/numBuckets;

    cudaDoHistogramPopulation_kernel<<<gridSize, blockSize>>>(d_input, d_output,
                                                              numElements,
                                                              bucketSize);

    hipMemcpy(h_outputHistogram, d_output, sizeof(unsigned int) * numBuckets,
            hipMemcpyDeviceToHost);

}
