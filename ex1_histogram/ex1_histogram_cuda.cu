#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <cstdio>

#include <hip/hip_runtime.h>

#include "ex1_histogram_cuda.cuh"

__global__
void
cudaDoHistogramPopulation_kernel(unsigned int * d_input, unsigned int * d_output,
                                unsigned int numElements,
                                unsigned int bucketSize) {
  unsigned int myID = (blockIdx.x * blockDim.x) + threadIdx.x;

  if(myID < numElements) {
    const unsigned int value = d_input[myID];
    const unsigned int bucketNumber = value / bucketSize;
    d_output[bucketNumber] += 1;
  }
}

void
cudaDoHistogramPopulation(const unsigned int threadsPerBlock,
                          unsigned int * h_outputHistogram,
                          unsigned int * d_input,
                          unsigned int * d_output,
                          unsigned int numElements,
                          unsigned int numBuckets) {

    dim3 blockSize(threadsPerBlock);
    dim3 gridSize((numElements / threadsPerBlock) + 1);
    const unsigned int bucketSize = numElements/numBuckets;

    cudaDoHistogramPopulation_kernel<<<gridSize, blockSize>>>(d_input, d_output,
                                                              numElements,
                                                              bucketSize);

    hipMemcpy(h_outputHistogram, d_output, sizeof(unsigned int) * numBuckets,
            hipMemcpyDeviceToHost);

}
