#include "hip/hip_runtime.h"
// -*- C++ -*-
// matrixMultiplication.cc
// a huge comparison of doing naive and tiled matrix multiplication using many
//  different methods and technologies

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <string>
#include <algorithm>

// yucky, but for asking the system how many cores we have
#include <unistd.h>

// header file for openmp
#include <omp.h>

// header files for kokkos
#include <Kokkos_Core.hpp>
#include "Teuchos_Array.hpp"
#include "Intrepid_ArrayTools.hpp"
#include "Intrepid_FieldContainer.hpp"
#include "Intrepid_RealSpaceTools.hpp"
#include "Teuchos_oblackholestream.hpp"
#include "Teuchos_RCP.hpp"
#include "Teuchos_ScalarTraits.hpp"
#include "Teuchos_GlobalMPISession.hpp"

#include <hip/hip_runtime.h>

using std::string;
using std::vector;
using Intrepid::FieldContainer;

typedef Intrepid::RealSpaceTools<double> rst;

#define BLOCK_SIZE 64;

//Pre-C++11 timing (thanks jeff)
double getElapsedTime(const timespec start, const timespec end) {
	timespec temp;
	if ((end.tv_nsec-start.tv_nsec)<0) {
		temp.tv_sec = end.tv_sec-start.tv_sec-1;
		temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
	} else {
		temp.tv_sec = end.tv_sec-start.tv_sec;
		temp.tv_nsec = end.tv_nsec-start.tv_nsec;
	}
	return double(temp.tv_sec) + double(temp.tv_nsec) / 1e9;
}


__global__
void
cudaDoContractFieldFieldTensor_kernel(double * d_left, double * d_right,
		double * d_out,
		int numCells,
		int numLeftFields,
		int numRightFields,
		int numPoints,
		int dim1Tensor,
		int dim2Tensor) {

	int myID = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(myID < numCells) {
		for (int lbf = 0; lbf < numLeftFields; lbf++) {
			for (int rbf = 0; rbf < numRightFields; rbf++) {
				double tmpVal = 0;
				for (int qp = 0; qp < numPoints; qp++) {
					for (int iTens1 = 0; iTens1 < dim1Tensor; iTens1++) {
						for (int iTens2 = 0; iTens2 < dim2Tensor; iTens2++) {
							tmpVal +=
								d_left[lbf*numPoints*dim1Tensor*dim2Tensor*numCells +
								qp*dim1Tensor*dim2Tensor*numCells +
								iTens1*dim2Tensor*numCells + iTens2*numCells + myID]
								*d_right[rbf*numPoints*dim1Tensor*dim2Tensor*numCells +
								qp*dim1Tensor*dim2Tensor*numCells +
								iTens1*dim2Tensor*numCells + iTens2*numCells + myID];
						} // D2-loop
					} // D1-loop
				} // P-loop
				d_out[lbf*numRightFields*numCells +
					rbf*numCells + myID] = tmpVal;
			} // R-loop
		} // L-loop
	}
}

void
cudaDoContractFieldFieldTensor(double * h_out,
		double * h_inLeft,
		double * h_inRight,
		int numCells,
		int numLeftFields,
		int numRightFields,
		int numPoints,
		int dim1Tensor,
		int dim2Tensor) {

	double * d_right;
	double * d_left;
	double * d_out;

	hipMalloc(&d_right, sizeof(double) * numCells * numRightFields * numPoints *
			dim1Tensor * dim2Tensor);

	hipMalloc(&d_left, sizeof(double) * numCells * numLeftFields * numPoints *
			dim1Tensor * dim2Tensor);

	hipMalloc(&d_out, sizeof(double) * numCells * numLeftFields * numRightFields);

	hipMemset(d_out, 0, sizeof(double) * numCells * numLeftFields *
			numRightFields);

	hipMemcpy(d_right, h_inRight,
			sizeof(double) * numCells * numRightFields * numPoints *
			dim1Tensor * dim2Tensor, hipMemcpyHostToDevice);

	hipMemcpy(d_left, h_inLeft,
			sizeof(double) * numCells * numLeftFields * numPoints *
			dim1Tensor * dim2Tensor, hipMemcpyHostToDevice);


	dim3 blockSize(64);
	dim3 gridSize((numCells / 64) + 1);

	cudaDoContractFieldFieldTensor_kernel<<<gridSize, blockSize>>>(d_left,
			d_right, d_out, numCells, numLeftFields,numRightFields,numPoints,
			dim1Tensor, dim2Tensor);

	hipMemcpy(h_out, d_out, sizeof(double) * numCells * numLeftFields *
			numRightFields, hipMemcpyDeviceToHost);

}

template<class DeviceType, class LeftViewType, class RightViewType, class OutputViewType>
struct ContractFieldFieldTensorFunctorUnrolled {
	typedef DeviceType device_type;
	LeftViewType _leftFields;
	RightViewType _rightFields;
	OutputViewType _outputFields;
	int _numLeftFields;
	int _numRightFields;
	int _numPoints;
	int _dim1Tensor;
	int _dim2Tensor;

	ContractFieldFieldTensorFunctorUnrolled(LeftViewType leftFields,
			RightViewType rightFields,
			OutputViewType outputFields,
			int numLeftFields,
			int numRightFields,
			int numPoints,
			int dim1Tensor,
			int dim2Tensor) :
		_leftFields(leftFields),
		_rightFields(rightFields),
		_outputFields(outputFields),
		_numLeftFields(numLeftFields),
		_numRightFields(numRightFields),
		_numPoints(numPoints),
		_dim1Tensor(dim1Tensor),
		_dim2Tensor(dim2Tensor)
	{
		// Nothing to do
	}

	KOKKOS_INLINE_FUNCTION
		void operator()(const unsigned int elementIndex) const {
			int pointIndex = elementIndex / _numLeftFields;
			int leftField = elementIndex % _numLeftFields;

			for (int rbf = 0; rbf < _numRightFields; rbf++) {
				double tmpVal = 0;
				for (int qp = 0; qp < _numPoints; qp++) {
					for (int iTens1 = 0; iTens1 < _dim1Tensor; iTens1++) {
						for (int iTens2 = 0; iTens2 < _dim2Tensor; iTens2++) {
							tmpVal += _leftFields(pointIndex, leftField, qp, iTens1, iTens2)*_rightFields(pointIndex, rbf, qp, iTens1, iTens2);
						} // D2-loo
					} // D1-loop
				} // P-loop
				_outputFields(pointIndex, leftField, rbf) = tmpVal;
			} // R-loop
		}
};


template<class DeviceType, class LeftViewType, class RightViewType, class OutputViewType>
struct ContractFieldFieldTensorFunctor {
	typedef DeviceType device_type;
	LeftViewType _leftFields;
	RightViewType _rightFields;
	OutputViewType _outputFields;
	int _numLeftFields;
	int _numRightFields;
	int _numPoints;
	int _dim1Tensor;
	int _dim2Tensor;

	ContractFieldFieldTensorFunctor(LeftViewType leftFields,
			RightViewType rightFields,
			OutputViewType outputFields,
			int numLeftFields,
			int numRightFields,
			int numPoints,
			int dim1Tensor,
			int dim2Tensor) :
		_leftFields(leftFields),
		_rightFields(rightFields),
		_outputFields(outputFields),
		_numLeftFields(numLeftFields),
		_numRightFields(numRightFields),
		_numPoints(numPoints),
		_dim1Tensor(dim1Tensor),
		_dim2Tensor(dim2Tensor)
	{
		// Nothing to do
	}

	KOKKOS_INLINE_FUNCTION
		void operator()(const unsigned int elementIndex) const {

			for (int lbf = 0; lbf < _numLeftFields; lbf++) {
				for (int rbf = 0; rbf < _numRightFields; rbf++) {
					double tmpVal = 0;
					for (int qp = 0; qp < _numPoints; qp++) {
						for (int iTens1 = 0; iTens1 < _dim1Tensor; iTens1++) {
							for (int iTens2 = 0; iTens2 < _dim2Tensor; iTens2++) {
								tmpVal += _leftFields(elementIndex, lbf, qp, iTens1, iTens2)*_rightFields(elementIndex, rbf, qp, iTens1, iTens2);
							} // D2-loop
						} // D1-loop
					} // P-loop
					_outputFields(elementIndex, lbf, rbf) = tmpVal;
				} // R-loop
			} // L-loop
		}
};




template<class DeviceType, class LeftViewType, class RightViewType, class OutputViewType>
struct ContractFieldFieldTensorFunctor1D {
	typedef DeviceType device_type;
	LeftViewType _leftFields;
	RightViewType _rightFields;
	OutputViewType _outputFields;
	int _numLeftFields;
	int _numRightFields;
	int _numPoints;
	int _dim1Tensor;
	int _dim2Tensor;
	int _numCells;

	ContractFieldFieldTensorFunctor1D(LeftViewType leftFields,
			RightViewType rightFields,
			OutputViewType outputFields,
			int numLeftFields,
			int numRightFields,
			int numPoints,
			int dim1Tensor,
			int dim2Tensor,
			int numCells) :
		_leftFields(leftFields),
		_rightFields(rightFields),
		_outputFields(outputFields),
		_numLeftFields(numLeftFields),
		_numRightFields(numRightFields),
		_numPoints(numPoints),
		_dim1Tensor(dim1Tensor),
		_dim2Tensor(dim2Tensor),
		_numCells(numCells)
	{
		// Nothing to do
	}

	KOKKOS_INLINE_FUNCTION
		void operator()(const unsigned int elementIndex) const {

			for (int lbf = 0; lbf < _numLeftFields; lbf++) {
				for (int rbf = 0; rbf < _numRightFields; rbf++) {
					double tmpVal = 0;
					for (int qp = 0; qp < _numPoints; qp++) {
						for (int iTens1 = 0; iTens1 < _dim1Tensor; iTens1++) {
							for (int iTens2 = 0; iTens2 < _dim2Tensor; iTens2++) {
								tmpVal +=
									_leftFields(lbf*_numPoints*_dim1Tensor*_dim2Tensor*_numCells +
											qp*_dim1Tensor*_dim2Tensor*_numCells +
											iTens1*_dim2Tensor*_numCells + iTens2*_numCells + elementIndex)
									*_rightFields(rbf*_numPoints*_dim1Tensor*_dim2Tensor*_numCells +
											qp*_dim1Tensor*_dim2Tensor*_numCells +
											iTens1*_dim2Tensor*_numCells + iTens2*_numCells + elementIndex);
							} // D2-loop
						} // D1-loop
					} // P-loop
					_outputFields(lbf*_numRightFields*_numCells +
							rbf*_numCells + elementIndex) = tmpVal;
				} // R-loop
			} // L-loop
		}
};





// Serial contractFieldFieldTensor.  Contracts FieldContainers of doubles.
void contractFieldFieldTensorSerial(FieldContainer<double> &  outputFields,
		const FieldContainer<double> &              leftFields,
		const FieldContainer<double> &              rightFields,
		double *                                    time = 0) {

	// TODO(ellen): Might later want to template this so that both the container
	//              and the scalars inside the container are template arguments,
	//              so we can hand it kokkos views or custom structs.

	// get sizes
	int numCells        = leftFields.dimension(0);
	int numLeftFields   = leftFields.dimension(1);
	int numRightFields  = rightFields.dimension(1);
	int numPoints       = leftFields.dimension(2);
	int dim1Tensor      = leftFields.dimension(3);
	int dim2Tensor      = leftFields.dimension(4);

	for (int cl = 0; cl < numCells; cl++) {
		for (int lbf = 0; lbf < numLeftFields; lbf++) {
			for (int rbf = 0; rbf < numRightFields; rbf++) {
				double tmpVal = 0;
				for (int qp = 0; qp < numPoints; qp++) {
					for (int iTens1 = 0; iTens1 < dim1Tensor; iTens1++) {
						for (int iTens2 = 0; iTens2 < dim2Tensor; iTens2++) {
							tmpVal += leftFields(cl, lbf, qp, iTens1, iTens2) * rightFields(cl, rbf, qp, iTens1, iTens2);
						} // D2-loop
					} // D1-loop
				} // P-loop
				outputFields(cl, lbf, rbf) = tmpVal;
			} // R-loop
		} // L-loop
	} // C-loop
}

/*
 * Kokkos Cuda contractfieldfieldtensor.
 *
 * Contracts two Kokkos Cuda host views (two double ***** tensors -> one double
 * *** tensor). Since
 *
 * Note that all input and output is in Kokkos host views --- the user is
 * responsible for getting the data in and out of them.
 */
template <class DeviceType, class input_view_t, class output_view_t, class input_host_t, class output_host_t>
void contractFieldFieldTensorKokkos(output_host_t &   outHost,
		const input_host_t &                      leftHost,
		const input_host_t &                      rightHost,
		output_view_t &                           outDevice,
		input_view_t &                            leftDevice,
		input_view_t &                            rightDevice,
		double *                                  time = 0) {

	// get sizes
	int numCells        = leftHost.dimension(0);
	int numLeftFields   = leftHost.dimension(1);
	int numRightFields  = rightHost.dimension(1);
	int numPoints       = leftHost.dimension(2);
	int dim1Tensor      = leftHost.dimension(3);
	int dim2Tensor      = leftHost.dimension(4);

	// Deep copy Kokkos host views into device views
	Kokkos::deep_copy(leftDevice, leftHost);
	Kokkos::deep_copy(rightDevice, rightHost);
	Kokkos::deep_copy(outDevice, outHost);

	timespec tic;
	if(time != 0)
		clock_gettime(CLOCK_MONOTONIC, &tic);

	ContractFieldFieldTensorFunctor<DeviceType, input_view_t, input_view_t, output_view_t>
		kokkosFunctor(leftDevice, rightDevice, outDevice, numLeftFields,
				numRightFields, numPoints, dim1Tensor, dim2Tensor);

	Kokkos::parallel_for(numCells, kokkosFunctor);

	Kokkos::fence();

	timespec toc;
	if(time !=0){
		clock_gettime(CLOCK_MONOTONIC, &toc);
		*time += getElapsedTime(tic, toc);
	}

	Kokkos::deep_copy(outHost, outDevice);
}


template <class DeviceType, class input_view_t, class output_view_t, class input_host_t, class output_host_t>
void contractFieldFieldTensorKokkos1D(output_host_t &   outHost,
		const input_host_t &                      leftHost,
		const input_host_t &                      rightHost,
		output_view_t &                           outDevice,
		input_view_t &                            leftDevice,
		input_view_t &                            rightDevice,
		int   numCells,
		int numLeftFields,
		int numRightFields,
		int numPoints,
		int dim1Tensor,
		int dim2Tensor,
		double *                                  time = 0
		) {
	/*
	// get sizes
	int numCells        = leftHost.dimension(0);
	int numLeftFields   = leftHost.dimension(1);
	int numRightFields  = rightHost.dimension(1);
	int numPoints       = leftHost.dimension(2);
	int dim1Tensor      = leftHost.dimension(3);
	int dim2Tensor      = leftHost.dimension(4);
	 */


	// Deep copy Kokkos host views into device views
	Kokkos::deep_copy(leftDevice, leftHost);
	Kokkos::deep_copy(rightDevice, rightHost);
	Kokkos::deep_copy(outDevice, outHost);

	timespec tic;
	if(time != 0)
		clock_gettime(CLOCK_MONOTONIC, &tic);

	ContractFieldFieldTensorFunctor1D<DeviceType, input_view_t, input_view_t, output_view_t>
		kokkosFunctor(leftDevice, rightDevice, outDevice, numLeftFields,
				numRightFields, numPoints, dim1Tensor, dim2Tensor, numCells);

	Kokkos::parallel_for(numCells, kokkosFunctor);

	Kokkos::fence();

	timespec toc;
	if(time !=0){
		clock_gettime(CLOCK_MONOTONIC, &toc);
		*time += getElapsedTime(tic, toc);
	}

	Kokkos::deep_copy(outHost, outDevice);
}



int main(int argc, char* argv[]) {

	int c=5000, p=20, l=3, r=7, d1=13, d2=5;

	// Make some field containers
	FieldContainer<double> in_c_l_p_d_d(c, l, p, d1, d2);
	FieldContainer<double> in_c_r_p_d_d(c, r, p, d1, d2);
	FieldContainer<double> out1_c_l_r(c, l, r);
	FieldContainer<double> out2_c_l_r(c, l, r);
	double zero = Intrepid::INTREPID_TOL*10000.0;

	// Fill field containers with random numbers
	for (int i=0; i<in_c_l_p_d_d.size(); i++) {
		in_c_l_p_d_d[i] = Teuchos::ScalarTraits<double>::random();
	}
	for (int i=0; i<in_c_r_p_d_d.size(); i++) {
		in_c_r_p_d_d[i] = Teuchos::ScalarTraits<double>::random();
	}

	// ===============================================================
	// ********************** < Kokkos setup> ************************
	// vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv

	// Doing all of this here might throw off the timing -- we're not counting the
	// cost of the copy into Kokkos or the deep copy from Kokkos host to Kokkos
	// device.

	Kokkos::initialize();

	// Kokkos Cuda views
	typedef Kokkos::View<double*, Kokkos::Cuda> cuda_input_view_t;
	typedef Kokkos::View<double *, Kokkos::Cuda> cuda_input_view_t;
	typedef Kokkos::View<double *, Kokkos::Cuda> cuda_output_view_t;
	typedef typename cuda_input_view_t::HostMirror cuda_input_host_t;
	typedef typename cuda_output_view_t::HostMirror cuda_output_host_t;

	// Kokkos OpenMP views
	typedef Kokkos::View<double *, Kokkos::OpenMP> omp_input_view_t;
	typedef Kokkos::View<double *, Kokkos::OpenMP> omp_output_view_t;
	typedef typename omp_input_view_t::HostMirror omp_input_host_t;
	typedef typename omp_output_view_t::HostMirror omp_output_host_t;

	//Cuda arrays
	double * cudaRight = new double[c * r * p * d1 * d2];
	double * cudaLeft = new double[c * l * p * d1 * d2];
	double * cudaOut = new double[c * l * r];

	// Make equivalent Kokkos views
	cuda_input_view_t cuda_kokkosLeft("left_input", c * l * p * d1 * d2);
	cuda_input_view_t cuda_kokkosRight("right_input", c * r * p * d1 * d2);
	cuda_output_view_t cuda_kokkosOut("output", c * l * r);

	omp_input_view_t omp_kokkosLeft("left_input", c * l * p * d1 * d2);
	omp_input_view_t omp_kokkosRight("right_input",  c * r * p * d1 * d2);
	omp_output_view_t omp_kokkosOut("output", c * l * r);

	// And their host mirrors
	cuda_input_host_t cuda_hostLeft = Kokkos::create_mirror_view(cuda_kokkosLeft);
	cuda_input_host_t cuda_hostRight = Kokkos::create_mirror_view(cuda_kokkosRight);
	cuda_output_host_t cuda_hostOut = Kokkos::create_mirror_view(cuda_kokkosOut);

	omp_input_host_t omp_hostLeft = Kokkos::create_mirror_view(omp_kokkosLeft);
	omp_input_host_t omp_hostRight = Kokkos::create_mirror_view(omp_kokkosRight);
	omp_output_host_t omp_hostOut = Kokkos::create_mirror_view(omp_kokkosOut);

	// Copy into Kokkos host views and cuda
	// Need to change this so that its 1-D and cl has stride 1
	for (int cl = 0; cl < c; ++cl) {
		for (int lbf = 0; lbf < l; ++lbf) {
			for (int qp = 0; qp < p; ++qp) {
				for (int iTens1 = 0; iTens1 < d1; ++iTens1) {
					for (int iTens2 = 0; iTens2 < d2; ++iTens2) {
						cuda_hostLeft(lbf*p*d1*d2*c + qp*d1*d2*c + iTens1*d2*c + iTens2*c +
								cl) = in_c_l_p_d_d(cl, lbf, qp, iTens1, iTens2);
						omp_hostLeft(lbf*p*d1*d2*c + qp*d1*d2*c + iTens1*d2*c + iTens2*c +
								cl) = in_c_l_p_d_d(cl, lbf, qp, iTens1, iTens2);
						cudaLeft[lbf*p*d1*d2*c + qp*d1*d2*c + iTens1*d2*c + iTens2*c +
							cl] = in_c_l_p_d_d(cl, lbf, qp, iTens1, iTens2);
					}
				}
			}
		}
		for (int rbf = 0; rbf < r; ++rbf) {
			for (int qp = 0; qp < p; ++qp) {
				for (int iTens1 = 0; iTens1 < d1; ++iTens1) {
					for (int iTens2 = 0; iTens2 < d2; ++iTens2) {
						cuda_hostRight(rbf*p*d1*d2*c + qp*d1*d2*c + iTens1*d2*c + iTens2*c +
								cl) = in_c_r_p_d_d(cl, rbf, qp, iTens1, iTens2);
						omp_hostRight(rbf*p*d1*d2*c + qp*d1*d2*c + iTens1*d2*c + iTens2*c +
								cl) = in_c_r_p_d_d(cl, rbf, qp, iTens1, iTens2);
						cudaRight[rbf*p*d1*d2*c + qp*d1*d2*c + iTens1*d2*c + iTens2*c +
							cl] = in_c_r_p_d_d(cl, rbf, qp, iTens1, iTens2);
					}
				}
			}
		}
	}








#if 0
	// Kokkos Cuda views
	typedef Kokkos::View<double *****, Kokkos::Cuda> cuda_input_view_t;
	typedef Kokkos::View<double ***, Kokkos::Cuda> cuda_output_view_t;
	typedef typename cuda_input_view_t::HostMirror cuda_input_host_t;
	typedef typename cuda_output_view_t::HostMirror cuda_output_host_t;

	// Kokkos OpenMP views
	typedef Kokkos::View<double *****, Kokkos::OpenMP> omp_input_view_t;
	typedef Kokkos::View<double ***, Kokkos::OpenMP> omp_output_view_t;
	typedef typename omp_input_view_t::HostMirror omp_input_host_t;
	typedef typename omp_output_view_t::HostMirror omp_output_host_t;

	// Make equivalent Kokkos views
	cuda_input_view_t cuda_kokkosLeft("left_input", c, l, p, d1, d2);
	cuda_input_view_t cuda_kokkosRight("right_input", c, r, p, d1, d2);
	cuda_output_view_t cuda_kokkosOut("output", c, l, r);

	omp_input_view_t omp_kokkosLeft("left_input", c, l, p, d1, d2);
	omp_input_view_t omp_kokkosRight("right_input", c, r, p, d1, d2);
	omp_output_view_t omp_kokkosOut("output", c, l, r);

	// And their host mirrors
	cuda_input_host_t cuda_hostLeft = Kokkos::create_mirror_view(cuda_kokkosLeft);
	cuda_input_host_t cuda_hostRight = Kokkos::create_mirror_view(cuda_kokkosRight);
	cuda_output_host_t cuda_hostOut = Kokkos::create_mirror_view(cuda_kokkosOut);

	omp_input_host_t omp_hostLeft = Kokkos::create_mirror_view(omp_kokkosLeft);
	omp_input_host_t omp_hostRight = Kokkos::create_mirror_view(omp_kokkosRight);
	omp_output_host_t omp_hostOut = Kokkos::create_mirror_view(omp_kokkosOut);

	// Copy into Kokkos host views.
	for (int cl = 0; cl < c; ++cl) {
		for (int lbf = 0; lbf < l; ++lbf) {
			for (int qp = 0; qp < p; ++qp) {
				for (int iTens1 = 0; iTens1 < d1; ++iTens1) {
					for (int iTens2 = 0; iTens2 < d2; ++iTens2) {
						cuda_hostLeft(cl, lbf, qp, iTens1, iTens2) = in_c_l_p_d_d(cl, lbf, qp, iTens1, iTens2);
						omp_hostLeft(cl, lbf, qp, iTens1, iTens2) = in_c_l_p_d_d(cl, lbf, qp, iTens1, iTens2);
					}
				}
			}
		}

		for (int rbf = 0; rbf < r; ++rbf) {
			for (int qp = 0; qp < p; ++qp) {
				for (int iTens1 = 0; iTens1 < d1; ++iTens1) {
					for (int iTens2 = 0; iTens2 < d2; ++iTens2) {
						cuda_hostRight(cl, rbf, qp, iTens1, iTens2) = in_c_r_p_d_d(cl, rbf, qp, iTens1, iTens2);
						omp_hostRight(cl, rbf, qp, iTens1, iTens2) = in_c_r_p_d_d(cl, rbf, qp, iTens1, iTens2);
					}
				}
			}
		}
	}
#endif

	// ===============================================================
	// ********************** </Kokkos setup> ************************
	// vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv

	printf("trying serial\n");

	//Warmup
	contractFieldFieldTensorSerial(out2_c_l_r, in_c_l_p_d_d, in_c_r_p_d_d);

	timespec tic;
	clock_gettime(CLOCK_MONOTONIC, &tic);

	//repeat the calculation 5 times so we can average out some randomness
	for(int i = 0; i < 5; ++i){
		contractFieldFieldTensorSerial(out2_c_l_r, in_c_l_p_d_d, in_c_r_p_d_d);
	}

	timespec toc;
	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_serial = getElapsedTime(tic, toc);

	printf("trying kokkos openmp\n");


#if 0
	//Warmpup
	contractFieldFieldTensorKokkos<Kokkos::OpenMP, omp_input_view_t, omp_output_view_t, omp_input_host_t, omp_output_host_t>
		(omp_hostOut, omp_hostLeft, omp_hostRight, omp_kokkosOut,
		 omp_kokkosLeft,omp_kokkosRight); clock_gettime(CLOCK_MONOTONIC, &tic);

	//repeat the calculation 5 times so we can average out some randomness
	for(int i = 0; i < 5; ++i){
		contractFieldFieldTensorKokkos<Kokkos::OpenMP, omp_input_view_t, omp_output_view_t, omp_input_host_t, omp_output_host_t>
			(omp_hostOut, omp_hostLeft, omp_hostRight, omp_kokkosOut, omp_kokkosLeft,
			 omp_kokkosRight);
	}

	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_kokkos = getElapsedTime(tic, toc);

	// Copy out from kokkos output view (NOT timing this)
	for (int cl = 0; cl < c; ++cl) {
		for (int lbf = 0; lbf < l; ++lbf) {
			for (int rbf = 0; rbf < r; ++rbf) {
				out1_c_l_r(cl, lbf, rbf) = omp_hostOut(cl, lbf, rbf);
			}
		}
	}
#endif

	contractFieldFieldTensorKokkos1D<Kokkos::OpenMP, omp_input_view_t,
		omp_output_view_t, omp_input_host_t, omp_output_host_t>(omp_hostOut, omp_hostLeft, omp_hostRight, omp_kokkosOut,
				omp_kokkosLeft,omp_kokkosRight, c, l, r, p, d1, d2); clock_gettime(CLOCK_MONOTONIC, &tic);

	//repeat the calculation 5 times so we can average out some randomness
	for(int i = 0; i < 5; ++i){
		contractFieldFieldTensorKokkos1D<Kokkos::OpenMP, omp_input_view_t, omp_output_view_t, omp_input_host_t, omp_output_host_t>
			(omp_hostOut, omp_hostLeft, omp_hostRight, omp_kokkosOut, omp_kokkosLeft,
			 omp_kokkosRight, c, l, r, p, d1, d2);
	}

	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_kokkos = getElapsedTime(tic, toc);

	// Copy out from kokkos output view (NOT timing this)
	for (int cl = 0; cl < c; ++cl) {
		for (int lbf = 0; lbf < l; ++lbf) {
			for (int rbf = 0; rbf < r; ++rbf) {
				out1_c_l_r(cl, lbf, rbf) = omp_hostOut(lbf*r*c + rbf*c + cl);
			}
		}
	}





	rst::subtract(&out1_c_l_r[0], &out2_c_l_r[0], out2_c_l_r.size());
	if (rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (1): check COMP_CPP vs. COMP_KOKKOS; "
			<< " diff-1norm = " << rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) << "\n\n";
	}
	else {
		std::cout << "omp kokkos speedup of " << elapsedTime_serial/elapsedTime_kokkos << std::endl;
	}

	//Now try the kokkos version without the copying of things in/out
	double elapsedTime_kokkos_noCopy = 0;

	for(int i = 0; i < 5; ++i){
		contractFieldFieldTensorKokkos1D<Kokkos::OpenMP, omp_input_view_t, omp_output_view_t, omp_input_host_t, omp_output_host_t>
			(omp_hostOut, omp_hostLeft, omp_hostRight, omp_kokkosOut, omp_kokkosLeft,
			 omp_kokkosRight, c, l, r, p, d1, d2, &elapsedTime_kokkos_noCopy);
	}

	// Copy out from kokkos output view (NOT timing this)
	for (int cl = 0; cl < c; ++cl) {
		for (int lbf = 0; lbf < l; ++lbf) {
			for (int rbf = 0; rbf < r; ++rbf) {
				out1_c_l_r(cl, lbf, rbf) = omp_hostOut(lbf*r*c + rbf*c + cl);
			}
		}
	}

	rst::subtract(&out1_c_l_r[0], &out2_c_l_r[0], out2_c_l_r.size());
	if (rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (1): check COMP_CPP vs. COMP_KOKKOS; "
			<< " diff-1norm = " << rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) << "\n\n";
	}
	std::cout << "omp kokkos no-copy speedup of " << elapsedTime_serial/elapsedTime_kokkos_noCopy << std::endl;

	printf("trying kokkos cuda\n");


	//Warmpup
	contractFieldFieldTensorKokkos1D<Kokkos::Cuda, cuda_input_view_t,
		cuda_output_view_t, cuda_input_host_t, cuda_output_host_t>
			(cuda_hostOut, cuda_hostLeft, cuda_hostRight, cuda_kokkosOut,
			 cuda_kokkosLeft, cuda_kokkosRight, c, l, r, p, d1, d2);
	clock_gettime(CLOCK_MONOTONIC, &tic);

	//repeat the calculation 5 times so we can average out some randomness
	for(int i = 0; i < 5; ++i){
		contractFieldFieldTensorKokkos1D<Kokkos::Cuda, cuda_input_view_t, cuda_output_view_t, cuda_input_host_t, cuda_output_host_t>
			(cuda_hostOut, cuda_hostLeft, cuda_hostRight, cuda_kokkosOut,
			 cuda_kokkosLeft, cuda_kokkosRight, c, l, r, p, d1, d2);
	}

	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_kokkos_cuda = getElapsedTime(tic, toc);

	// Copy out from kokkos output view (NOT timing this)
	for (int cl = 0; cl < c; ++cl) {
		for (int lbf = 0; lbf < l; ++lbf) {
			for (int rbf = 0; rbf < r; ++rbf) {
				out1_c_l_r(cl, lbf, rbf) = omp_hostOut(lbf*r*c + rbf*c + cl);
			}
		}
	}

	rst::subtract(&out1_c_l_r[0], &out2_c_l_r[0], out2_c_l_r.size());
	if (rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (1): check COMP_CPP vs. COMP_KOKKOS; "
			<< " diff-1norm = " << rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) << "\n\n";
	}

	std::cout << "cuda kokkos speedup of " << elapsedTime_serial/elapsedTime_kokkos_cuda << std::endl;

	//Now try the kokkos version without the copying of things in/out
	double elapsedTime_kokkos_noCopyCuda = 0;

	for(int i = 0; i < 5; ++i){
		contractFieldFieldTensorKokkos1D<Kokkos::Cuda, cuda_input_view_t, cuda_output_view_t, cuda_input_host_t, cuda_output_host_t>
			(cuda_hostOut, cuda_hostLeft, cuda_hostRight, cuda_kokkosOut,
			 cuda_kokkosLeft, cuda_kokkosRight, c, l, r, p, d1, d2, &elapsedTime_kokkos_noCopyCuda);
	}

	for (int cl = 0; cl < c; ++cl) {
		for (int lbf = 0; lbf < l; ++lbf) {
			for (int rbf = 0; rbf < r; ++rbf) {
				out1_c_l_r(cl, lbf, rbf) = omp_hostOut(lbf*r*c + rbf*c + cl);
			}
		}
	}

	rst::subtract(&out1_c_l_r[0], &out2_c_l_r[0], out2_c_l_r.size());
	if (rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (1): check COMP_CPP vs. COMP_KOKKOS; "
			<< " diff-1norm = " << rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) << "\n\n";
	}

	std::cout << "cuda kokkos no-copy speedup of " << elapsedTime_serial/elapsedTime_kokkos_noCopyCuda << std::endl;

	Kokkos::finalize();

	//Now try the cuda version, start with warmup
	cudaDoContractFieldFieldTensor(cudaOut,cudaLeft,cudaRight, c, l, r,
			p, d1, d2);

	clock_gettime(CLOCK_MONOTONIC, &tic);
	for(int i = 0; i < 5; ++i){
		cudaDoContractFieldFieldTensor(cudaOut,cudaLeft,cudaRight, c, l, r,
				p, d1, d2);
	}

	clock_gettime(CLOCK_MONOTONIC, &toc);
	const double elapsedTime_cuda = getElapsedTime(tic, toc);

	for (int cl = 0; cl < c; ++cl) {
		for (int lbf = 0; lbf < l; ++lbf) {
			for (int rbf = 0; rbf < r; ++rbf) {
				out1_c_l_r(cl, lbf, rbf) = cudaOut[lbf*r*c + rbf*c + cl];
			}
		}
	}

	rst::subtract(&out1_c_l_r[0], &out2_c_l_r[0], out2_c_l_r.size());
	if (rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) > zero) {
		std::cout << "\n\nINCORRECT contractFieldFieldTensor (1): check cuda; "
			<< " diff-1norm = " << rst::vectorNorm(&out1_c_l_r[0], out1_c_l_r.size(), Intrepid::NORM_ONE) << "\n\n";
	}

	std::cout << "cuda speedup of " << elapsedTime_serial/elapsedTime_cuda << std::endl;


	return 0;
}
