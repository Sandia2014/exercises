#include "hip/hip_runtime.h"
// -*- C++ -*-
// matrixMultiplication_kokkos.cu
// kokkos gets its own file because it doesn't play nicely with the other things

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <vector>

// header files for kokkos
#include <Kokkos_Core.hpp>

using std::string;
using std::vector;

enum KokkosDeepCopyStyle {KokkosDoDeepCopiesEveryRepeat,
                          KokkosDontDoDeepCopiesEveryRepeat};

// yay for having to use pre-c++11 timing.
double
getElapsedTime(const timespec start, const timespec end) {
  timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return double(temp.tv_sec) + double(temp.tv_nsec) / 1e9;
}

template <class DeviceType, class KokkosLeftMatrix, class KokkosRightMatrix>
struct KokkosFunctor {

  typedef DeviceType device_type;

  const unsigned int _matrixSize;
  KokkosLeftMatrix _leftMatrix;
  KokkosRightMatrix _rightMatrix;
  KokkosLeftMatrix _resultMatrix;

  KokkosFunctor(const unsigned int matrixSize,
                KokkosLeftMatrix leftMatrix,
                KokkosRightMatrix rightMatrix,
                KokkosLeftMatrix resultMatrix) :
    _matrixSize(matrixSize), _leftMatrix(leftMatrix), _rightMatrix(rightMatrix),
    _resultMatrix(resultMatrix) {
  }

  KOKKOS_INLINE_FUNCTION
  void operator()(const unsigned int elementIndex) const {
    const unsigned int row = elementIndex / _matrixSize;
    const unsigned int col = elementIndex - row * _matrixSize;
    double sum = 0;
    for (unsigned int dummy = 0; dummy < _matrixSize; ++dummy) {
      sum += _leftMatrix(row, dummy) * _rightMatrix(dummy, col);
    }
    _resultMatrix(row, col) = sum;
  }

private:
  KokkosFunctor();

};

template <class DeviceType, class KokkosLeftMatrix, class KokkosRightMatrix>
double
runKokkosTest(const unsigned int matrixSize,
              const double cacheUnfriendlyCheckSum,
              const unsigned int numberOfRepeats,
              const vector<double> & leftMatrix,
              const vector<double> & rightMatrix,
              const KokkosDeepCopyStyle kokkosDeepCopyStyle) {

  typedef typename KokkosLeftMatrix::HostMirror   KokkosLeftMatrix_Host;
  typedef typename KokkosRightMatrix::HostMirror  KokkosRightMatrix_Host;

  // TODO: make device views for left, right, and result
  // TODO: make host views for left, right, and result

  // TODO: copy contents of leftMatrix and rightMatrix into device views

  // TODO: make a kokkos functor

  // (optional) warm up kokkos

  // start timing
  timespec tic;
  clock_gettime(CLOCK_MONOTONIC, &tic);

  for (unsigned int repeatIndex = 0;
       repeatIndex < numberOfRepeats; ++repeatIndex) {

    // (optional) copy left and right matrices to device?

    // TODO: do the multiplication with kokkos
    // TODO: wait for the multiplication to finish

    // (optional) copy result view back to host?

  }

  // stop timing
  timespec toc;
  clock_gettime(CLOCK_MONOTONIC, &toc);
  const double elapsedTime = getElapsedTime(tic, toc);

  // compute checksum
  double checkSum = 0;
  // TODO: do you need to copy result matrix to host?
  for (unsigned int row = 0; row < matrixSize; ++row) {
    for (unsigned int col = 0; col < matrixSize; ++col) {
      //checkSum += // TODO: something
    }
  }
  printf("checkSum is %lf\n", checkSum);
  if (std::abs(cacheUnfriendlyCheckSum - checkSum) / cacheUnfriendlyCheckSum > 1e-6) {
    fprintf(stderr, "incorrect checksum = %lf, correct is %lf\n",
            checkSum, cacheUnfriendlyCheckSum);
    exit(1);
  }

  return elapsedTime;
}

int main(int argc, char* argv[]) {

  // a couple of inputs.  change the numberOfIntervals to control the amount
  //  of work done
  const unsigned int matrixSize = 512 * 3;
  const unsigned int numberOfRepeats = 1;

  printf("using a matrix size of %u\n", matrixSize);
  char methodName[500];

  vector<double> leftMatrix(matrixSize * matrixSize);
  vector<double> rightMatrix(matrixSize * matrixSize);
  vector<double> resultMatrix(matrixSize * matrixSize);
  for (unsigned int row = 0; row < matrixSize; ++row) {
    for (unsigned int col = 0; col < matrixSize; ++col) {
      leftMatrix[row * matrixSize + col] = rand() / double(RAND_MAX);
      rightMatrix[row * matrixSize + col] = rand() / double(RAND_MAX);
    }
  }

  // ===============================================================
  // ********************** < do cache unfriendly> *****************
  // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv

  timespec tic;
  clock_gettime(CLOCK_MONOTONIC, &tic);
  for (unsigned int repeatIndex = 0;
       repeatIndex < numberOfRepeats; ++repeatIndex) {
    for (unsigned int row = 0; row < matrixSize; ++row) {
      for (unsigned int col = 0; col < matrixSize; ++col) {
        double sum = 0;
        for (unsigned int dummy = 0; dummy < matrixSize; ++dummy) {
          sum +=
            leftMatrix[row * matrixSize + dummy] *
            rightMatrix[dummy * matrixSize + col];
        }
        resultMatrix[row * matrixSize + col] = sum;
      }
    }
  }
  timespec toc;
  clock_gettime(CLOCK_MONOTONIC, &toc);
  const double cacheUnfriendlyElapsedTime = getElapsedTime(tic, toc);

  double cacheUnfriendlyCheckSum = 0;
  for (unsigned int row = 0; row < matrixSize; ++row) {
    for (unsigned int col = 0; col < matrixSize; ++col) {
      cacheUnfriendlyCheckSum += resultMatrix[row * matrixSize + col];
    }
  }
  printf("%-38s : time %6.2f seconds\n",
         "cache unfriendly", cacheUnfriendlyElapsedTime);

  // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
  // ********************** </do cache unfriendly> *****************
  // ===============================================================

  Kokkos::initialize();
  // ===============================================================
  // ********************** < do kokkos> ***************************
  // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv

  const KokkosDeepCopyStyle kokkosDeepCopyStyle =
    KokkosDontDoDeepCopiesEveryRepeat;

  {
    typedef Kokkos::Cuda                           DeviceType;
    typedef Kokkos::View<double**, DeviceType>     KokkosMatrix;

    const double elapsedTime =
      runKokkosTest<DeviceType,
                    KokkosMatrix,
                    KokkosMatrix>(matrixSize,
                                  cacheUnfriendlyCheckSum,
                                  numberOfRepeats,
                                  leftMatrix,
                                  rightMatrix,
                                  kokkosDeepCopyStyle);

    sprintf(methodName, "naive kokkos cuda, %s",
            kokkosDeepCopyStyle == KokkosDoDeepCopiesEveryRepeat ? "deep copies" : "");
    printf("%-38s : time %6.2f speedup w.r.t. unfriendly %6.2f\n",
           methodName,
           elapsedTime,
           cacheUnfriendlyElapsedTime / elapsedTime);

  }

  {

    typedef Kokkos::OpenMP                         DeviceType;
    typedef Kokkos::View<double**, DeviceType>     KokkosMatrix;

    const double elapsedTime =
      runKokkosTest<DeviceType,
                    KokkosMatrix,
                    KokkosMatrix>(matrixSize,
                                  cacheUnfriendlyCheckSum,
                                  numberOfRepeats,
                                  leftMatrix,
                                  rightMatrix,
                                  kokkosDeepCopyStyle);

    sprintf(methodName, "naive kokkos omp %s",
            kokkosDeepCopyStyle == KokkosDoDeepCopiesEveryRepeat ? "deep copies" : "");
    printf("%-38s : time %6.2f speedup w.r.t. unfriendly %6.2f\n",
           methodName,
           elapsedTime,
           cacheUnfriendlyElapsedTime / elapsedTime);
  }

  {

    typedef Kokkos::OpenMP                               DeviceType;
    typedef Kokkos::View<double**, Kokkos::LayoutRight>  KokkosLeftMatrix;
    typedef Kokkos::View<double**, Kokkos::LayoutLeft>   KokkosRightMatrix;

    const double elapsedTime =
      runKokkosTest<DeviceType,
                    KokkosLeftMatrix,
                    KokkosRightMatrix>(matrixSize,
                                       cacheUnfriendlyCheckSum,
                                       numberOfRepeats,
                                       leftMatrix,
                                       rightMatrix,
                                       kokkosDeepCopyStyle);

    sprintf(methodName, "naive kokkos omp spec %s",
            kokkosDeepCopyStyle == KokkosDoDeepCopiesEveryRepeat ? "deep copies" : "");
    printf("%-38s : time %6.2f speedup w.r.t. unfriendly %6.2f\n",
           methodName,
           elapsedTime,
           cacheUnfriendlyElapsedTime / elapsedTime);

  }

  {

    typedef Kokkos::OpenMP                                           DeviceType;
    typedef Kokkos::View<double**, DeviceType, Kokkos::LayoutRight>  KokkosLeftMatrix;
    typedef Kokkos::View<double**, DeviceType, Kokkos::LayoutLeft>   KokkosRightMatrix;

    const double elapsedTime =
      runKokkosTest<DeviceType,
                    KokkosLeftMatrix,
                    KokkosRightMatrix>(matrixSize,
                                       cacheUnfriendlyCheckSum,
                                       numberOfRepeats,
                                       leftMatrix,
                                       rightMatrix,
                                       kokkosDeepCopyStyle);

    sprintf(methodName, "naive kokkos omp spec broken %s",
            kokkosDeepCopyStyle == KokkosDoDeepCopiesEveryRepeat ? "deep copies" : "");
    printf("%-38s : time %6.2f speedup w.r.t. unfriendly %6.2f\n",
           methodName,
           elapsedTime,
           cacheUnfriendlyElapsedTime / elapsedTime);

  }

  // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
  // ********************** </do kokkos> ***************************
  // ===============================================================
  Kokkos::finalize();

  return 0;
}
