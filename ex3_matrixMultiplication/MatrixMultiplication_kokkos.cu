#include "hip/hip_runtime.h"
// -*- C++ -*-
// matrixMultiplication_kokkos.cu
// kokkos gets its own file because it doesn't play nicely with the other things

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <vector>

// header files for kokkos
#include <Kokkos_Core.hpp>

using std::string;
using std::vector;
typedef Kokkos::View<double *> matrixView_type;
typedef matrixView_type::HostMirror host_matrix;

enum KokkosDeepCopyStyle {KokkosDoDeepCopiesEveryRepeat,
                          KokkosDontDoDeepCopiesEveryRepeat};

// yay for having to use pre-c++11 timing.
double
getElapsedTime(const timespec start, const timespec end) {
  timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return double(temp.tv_sec) + double(temp.tv_nsec) / 1e9;
}

template <class DeviceType, class KokkosLeftMatrix, class KokkosRightMatrix>
struct KokkosFunctor {

  typedef DeviceType device_type;

  const unsigned int _matrixSize;
  KokkosLeftMatrix _leftMatrix;
  KokkosRightMatrix _rightMatrix;
  KokkosLeftMatrix _resultMatrix;

  KokkosFunctor(const unsigned int matrixSize,
                KokkosLeftMatrix leftMatrix,
                KokkosRightMatrix rightMatrix,
                KokkosLeftMatrix resultMatrix) :
    _matrixSize(matrixSize), _leftMatrix(leftMatrix), _rightMatrix(rightMatrix),
    _resultMatrix(resultMatrix) {
  }

  KOKKOS_INLINE_FUNCTION
  void operator()(const unsigned int elementIndex) const {
    const unsigned int row = elementIndex / _matrixSize;
    const unsigned int col = elementIndex - row * _matrixSize;
    double sum = 0;
    for (unsigned int dummy = 0; dummy < _matrixSize; ++dummy) {
      sum += _leftMatrix(row, dummy) * _rightMatrix(dummy, col);
    }
    _resultMatrix(row, col) = sum;
  }

private:
  KokkosFunctor();

};

template <class DeviceType, class KokkosLeftMatrix, class KokkosRightMatrix>
double
runKokkosTest(const unsigned int matrixSize,
              const double cacheUnfriendlyCheckSum,
              const unsigned int numberOfRepeats,
              const vector<double> & leftMatrix,
              const vector<double> & rightMatrix,
              const KokkosDeepCopyStyle kokkosDeepCopyStyle) {

  typedef typename KokkosLeftMatrix::HostMirror   KokkosLeftMatrix_Host;
  typedef typename KokkosRightMatrix::HostMirror  KokkosRightMatrix_Host;

  // (optional) warm up kokkos
  KokkosLeftMatrix left("left", matrixSize, matrixSize);
  KokkosRightMatrix right("right", matrixSize, matrixSize);
  KokkosLeftMatrix warmup("warmup", matrixSize, matrixSize);
  KokkosLeftMatrix result("result", matrixSize, matrixSize);

  KokkosLeftMatrix_Host h_left = Kokkos::create_mirror_view(left);
  KokkosRightMatrix_Host h_right = Kokkos::create_mirror_view(right);
  KokkosLeftMatrix_Host h_warmup = Kokkos::create_mirror_view(warmup);
  KokkosLeftMatrix_Host h_result = Kokkos::create_mirror_view(result);

  for(unsigned row = 0; row < matrixSize; ++row) {
    for(unsigned col = 0; col < matrixSize; ++col) {
      h_left(row, col) = leftMatrix[row * matrixSize + col];
      h_right(row, col) = rightMatrix[row * matrixSize + col];
      h_result(row, col) = 0;
      h_warmup(row, col) = 0;
    }
  }

  Kokkos::deep_copy(left, h_left);
  Kokkos::deep_copy(right, h_right);
  Kokkos::deep_copy(result, h_result);
  Kokkos::deep_copy(warmup, h_warmup);

  KokkosFunctor<DeviceType, KokkosLeftMatrix, KokkosRightMatrix> kokkosFunctor
                                            (matrixSize, left, right, result);

  KokkosFunctor<DeviceType, KokkosLeftMatrix, KokkosRightMatrix> warmupFunctor
                                            (matrixSize, left, right, warmup);

  Kokkos::parallel_for(matrixSize*matrixSize, warmupFunctor);

  // start timing
  timespec tic;
  clock_gettime(CLOCK_MONOTONIC, &tic);

  for (unsigned int repeatIndex = 0;
       repeatIndex < numberOfRepeats; ++repeatIndex) {

    Kokkos::parallel_for(matrixSize*matrixSize, kokkosFunctor);
    Kokkos::fence();
    Kokkos::deep_copy(h_result, result);
  }

  // stop timing
  timespec toc;
  clock_gettime(CLOCK_MONOTONIC, &toc);
  const double elapsedTime = getElapsedTime(tic, toc);

  // compute checksum
  double checkSum = 0;
  // TODO: do you need to copy result matrix to host?
  for (unsigned int row = 0; row < matrixSize; ++row) {
    for (unsigned int col = 0; col < matrixSize; ++col) {
      checkSum += h_result(row, col);
    }
  }
  printf("checkSum is %lf\n", checkSum);
  if (std::abs(cacheUnfriendlyCheckSum - checkSum) / cacheUnfriendlyCheckSum > 1e-6) {
    fprintf(stderr, "incorrect checksum = %lf, correct is %lf\n",
            checkSum, cacheUnfriendlyCheckSum);
    exit(1);
  }

  return elapsedTime;
}

int main(int argc, char* argv[]) {

  // a couple of inputs.  change the numberOfIntervals to control the amount
  //  of work done
  const unsigned int matrixSize = 512 * 3;
  const unsigned int numberOfRepeats = 1;

  printf("using a matrix size of %u\n", matrixSize);
  char methodName[500];

  vector<double> leftMatrix(matrixSize * matrixSize);
  vector<double> rightMatrix(matrixSize * matrixSize);
  vector<double> resultMatrix(matrixSize * matrixSize);
  for (unsigned int row = 0; row < matrixSize; ++row) {
    for (unsigned int col = 0; col < matrixSize; ++col) {
      leftMatrix[row * matrixSize + col] = rand() / double(RAND_MAX);
      rightMatrix[row * matrixSize + col] = rand() / double(RAND_MAX);
    }
  }

  // ===============================================================
  // ********************** < do cache unfriendly> *****************
  // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv

  timespec tic;
  clock_gettime(CLOCK_MONOTONIC, &tic);
  for (unsigned int repeatIndex = 0;
       repeatIndex < numberOfRepeats; ++repeatIndex) {
    for (unsigned int row = 0; row < matrixSize; ++row) {
      for (unsigned int col = 0; col < matrixSize; ++col) {
        double sum = 0;
        for (unsigned int dummy = 0; dummy < matrixSize; ++dummy) {
          sum +=
            leftMatrix[row * matrixSize + dummy] *
            rightMatrix[dummy * matrixSize + col];
        }
        resultMatrix[row * matrixSize + col] = sum;
      }
    }
  }
  timespec toc;
  clock_gettime(CLOCK_MONOTONIC, &toc);
  const double cacheUnfriendlyElapsedTime = getElapsedTime(tic, toc);

  double cacheUnfriendlyCheckSum = 0;
  for (unsigned int row = 0; row < matrixSize; ++row) {
    for (unsigned int col = 0; col < matrixSize; ++col) {
      cacheUnfriendlyCheckSum += resultMatrix[row * matrixSize + col];
    }
  }
  printf("%-38s : time %6.2f seconds\n",
         "cache unfriendly", cacheUnfriendlyElapsedTime);

  // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
  // ********************** </do cache unfriendly> *****************
  // ===============================================================

  Kokkos::initialize();
  // ===============================================================
  // ********************** < do kokkos> ***************************
  // vvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvvv

  const KokkosDeepCopyStyle kokkosDeepCopyStyle =
    KokkosDontDoDeepCopiesEveryRepeat;

  {
    typedef Kokkos::Cuda                           DeviceType;
    typedef Kokkos::View<double**, DeviceType>     KokkosMatrix;

    const double elapsedTime =
      runKokkosTest<DeviceType,
                    KokkosMatrix,
                    KokkosMatrix>(matrixSize,
                                  cacheUnfriendlyCheckSum,
                                  numberOfRepeats,
                                  leftMatrix,
                                  rightMatrix,
                                  kokkosDeepCopyStyle);

    sprintf(methodName, "naive kokkos cuda, %s",
            kokkosDeepCopyStyle == KokkosDoDeepCopiesEveryRepeat ? "deep copies" : "");
    printf("%-38s : time %6.2f speedup w.r.t. unfriendly %6.2f\n",
           methodName,
           elapsedTime,
           cacheUnfriendlyElapsedTime / elapsedTime);

  }

  {

    typedef Kokkos::OpenMP                         DeviceType;
    typedef Kokkos::View<double**, DeviceType>     KokkosMatrix;

    const double elapsedTime =
      runKokkosTest<DeviceType,
                    KokkosMatrix,
                    KokkosMatrix>(matrixSize,
                                  cacheUnfriendlyCheckSum,
                                  numberOfRepeats,
                                  leftMatrix,
                                  rightMatrix,
                                  kokkosDeepCopyStyle);

    sprintf(methodName, "naive kokkos omp %s",
            kokkosDeepCopyStyle == KokkosDoDeepCopiesEveryRepeat ? "deep copies" : "");
    printf("%-38s : time %6.2f speedup w.r.t. unfriendly %6.2f\n",
           methodName,
           elapsedTime,
           cacheUnfriendlyElapsedTime / elapsedTime);
  }

  {

    typedef Kokkos::OpenMP                               DeviceType;
    typedef Kokkos::View<double**, Kokkos::LayoutRight>  KokkosLeftMatrix;
    typedef Kokkos::View<double**, Kokkos::LayoutLeft>   KokkosRightMatrix;

    const double elapsedTime =
      runKokkosTest<DeviceType,
                    KokkosLeftMatrix,
                    KokkosRightMatrix>(matrixSize,
                                       cacheUnfriendlyCheckSum,
                                       numberOfRepeats,
                                       leftMatrix,
                                       rightMatrix,
                                       kokkosDeepCopyStyle);

    sprintf(methodName, "naive kokkos omp spec %s",
            kokkosDeepCopyStyle == KokkosDoDeepCopiesEveryRepeat ? "deep copies" : "");
    printf("%-38s : time %6.2f speedup w.r.t. unfriendly %6.2f\n",
           methodName,
           elapsedTime,
           cacheUnfriendlyElapsedTime / elapsedTime);

  }

  {

    typedef Kokkos::OpenMP                                           DeviceType;
    typedef Kokkos::View<double**, DeviceType, Kokkos::LayoutRight>  KokkosLeftMatrix;
    typedef Kokkos::View<double**, DeviceType, Kokkos::LayoutLeft>   KokkosRightMatrix;

    const double elapsedTime =
      runKokkosTest<DeviceType,
                    KokkosLeftMatrix,
                    KokkosRightMatrix>(matrixSize,
                                       cacheUnfriendlyCheckSum,
                                       numberOfRepeats,
                                       leftMatrix,
                                       rightMatrix,
                                       kokkosDeepCopyStyle);

    sprintf(methodName, "naive kokkos omp spec broken %s",
            kokkosDeepCopyStyle == KokkosDoDeepCopiesEveryRepeat ? "deep copies" : "");
    printf("%-38s : time %6.2f speedup w.r.t. unfriendly %6.2f\n",
           methodName,
           elapsedTime,
           cacheUnfriendlyElapsedTime / elapsedTime);

  }

  // ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
  // ********************** </do kokkos> ***************************
  // ===============================================================
  Kokkos::finalize();

  return 0;
}
