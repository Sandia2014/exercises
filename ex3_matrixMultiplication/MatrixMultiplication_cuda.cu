#include "hip/hip_runtime.h"
// -*- C++ -*-
#include <cstdio>

#include <hip/hip_runtime.h>
// These come from the cublas matrix multiplication example
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wswitch"
#include <hipblas.h>
#pragma GCC diagnostic pop


#include "MatrixMultiplication_cuda.cuh"

__global__
void
cudaDoNaiveMatrixMultiplication_kernel(const unsigned int matrixSize,
                                       const double * leftMatrix,
                                       const double * rightMatrix,
                                       double * resultMatrix) {
  // TODO: something!
}

void
cudaDoMatrixMultiplication(const unsigned int maxNumberOfBlocks,
                           const unsigned int numberOfThreadsPerBlock,
                           const unsigned int matrixSize) {

  // TODO: something!
}


void
multiplyMatricesUsingCublas(const unsigned int matrixSize,
                            const double * leftMatrix,
                            const double * rightMatrix,
                            double * resultMatrix) {

  const unsigned int numberOfEntries = matrixSize * matrixSize;

  // allocate device memory
  double * dev_leftMatrix;
  double * dev_rightMatrix;
  double * dev_resultMatrix;
  hipMalloc((void **) &dev_leftMatrix, numberOfEntries * sizeof(double));
  hipMalloc((void **) &dev_rightMatrix, numberOfEntries * sizeof(double));
  hipMalloc((void **) &dev_resultMatrix, numberOfEntries * sizeof(double));
  // copy matrices to the device
  hipMemcpy(dev_leftMatrix, leftMatrix, numberOfEntries * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_rightMatrix, rightMatrix, numberOfEntries * sizeof(double), hipMemcpyHostToDevice);

  const double alpha = 1.0f;
  const double beta  = 0.0f;
  hipblasHandle_t handle;

  hipblasCreate(&handle);

  // perform the multiply
  hipblasDgemm(handle,
              HIPBLAS_OP_N, HIPBLAS_OP_N, // don't transpose
              matrixSize, matrixSize, matrixSize, // sizes
              &alpha, // no scalar premultiply
              dev_rightMatrix, matrixSize, // left matrix
              dev_leftMatrix, matrixSize, // right matrix
              &beta, // don't premultiply result by anything
              dev_resultMatrix, matrixSize);

  // copy result from device to host
  hipMemcpy(resultMatrix, dev_resultMatrix,
             numberOfEntries * sizeof(double),
             hipMemcpyDeviceToHost);

  // Destroy the handle
  hipblasDestroy(handle);

  // clean up memory
  hipFree(dev_leftMatrix);
  hipFree(dev_rightMatrix);
  hipFree(dev_resultMatrix);
}
